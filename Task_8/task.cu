#include "hip/hip_runtime.h"
#include <iostream>
#include <boost/program_options.hpp>
#include <cmath>
#include <memory>
#include <algorithm>
#include <fstream>
#include <iomanip>
#include <chrono>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

namespace po = boost::program_options;

#define at(arr, x, y) (arr[(x) * size + (y)])
#define size_sq size * size

constexpr int LEFT_UP = 10;
constexpr int LEFT_DOWN = 20;
constexpr int RIGHT_UP = 20;
constexpr int RIGHT_DOWN = 30;

template <class ctype>
class Data {
private:
    int len;
    ctype* d_arr;

public:
    std::vector<ctype> arr;

    Data(int length) : len(length), arr(len), d_arr(nullptr) {
        hipError_t err = hipMalloc((void**)&d_arr, len * sizeof(ctype));
        if (err != hipSuccess) {
            std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(err) << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    ~Data() {
        if (d_arr) {
            hipFree(d_arr);
        }
    }

    void copyToDevice() {
        hipError_t err = hipMemcpy(d_arr, arr.data(), len * sizeof(ctype), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "CUDA memory copy to device failed: " << hipGetErrorString(err) << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    void copyToHost() {
        hipError_t err = hipMemcpy(arr.data(), d_arr, len * sizeof(ctype), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "CUDA memory copy to host failed: " << hipGetErrorString(err) << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    ctype* getDevicePointer() {
        return d_arr;
    }
};


void initMatrix(std::vector<double>& mainArr, int size) {
    at(mainArr, 0, 0) = LEFT_UP;
    at(mainArr, 0, size - 1) = RIGHT_UP;
    at(mainArr, size - 1, 0) = LEFT_DOWN;
    at(mainArr, size - 1, size - 1) = RIGHT_DOWN;

    for (int i = 1; i < size - 1; i++)
    {
        at(mainArr, 0, i) = (at(mainArr, 0, size - 1) - at(mainArr, 0, 0)) / (size - 1) * i + at(mainArr, 0, 0);
        at(mainArr, i, 0) = (at(mainArr, size - 1, 0) - at(mainArr, 0, 0)) / (size - 1) * i + at(mainArr, 0, 0);

        at(mainArr, size - 1, i) = (at(mainArr, size - 1, size - 1) - at(mainArr, size - 1, 0)) / (size - 1) * i + at(mainArr, size - 1, 0);
        at(mainArr, i, size - 1) = (at(mainArr, size - 1, size - 1) - at(mainArr, 0, size - 1)) / (size - 1) * i + at(mainArr, 0, size - 1);
    }
}

void saveMatrix(const double* mainArr, int size, const std::string& filename) 
{
    std::ofstream outputFile(filename);
    if (!outputFile.is_open()) 
    {
        std::cerr << "Unable to open file " << filename << " for writing." << std::endl;
        return;
    }

    for (int i = 0; i < size; ++i) 
    {
        for (int j = 0; j < size; ++j) 
        {
            outputFile << std::setw(4) << std::fixed << std::setprecision(4) << at(mainArr, i, j) << ' ';
        }
        outputFile << std::endl;
    }
    outputFile.close();
}

__global__ void iterate(double* matrix, double* lastMatrix, int size) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j == 0 || i == 0 || i >= size - 1 || j >= size - 1) return;

    at(matrix, i, j) = 0.25 * (at(lastMatrix, i, j + 1) + at(lastMatrix, i, j - 1) +
                                at(lastMatrix, i - 1, j) + at(lastMatrix, i + 1, j));
}

template <unsigned int blockSize>
__global__ void compute_error(double* matrix, double* lastMatrix, double* errors, int size) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= size || i >= size) return;

    // Allocate the temporary storage for a block of blockSize(32) threads of type double
    __shared__ typename hipcub::BlockReduce<double, blockSize>::TempStorage temp_storage;
    double local_max = 0.0;

    if (j > 0 && i > 0 && j < size - 1 && i < size - 1) {
        local_max = fabs(at(matrix, i, j) - at(lastMatrix, i, j));
    }

    // Calculate the largest value in the block using the reduction operation
    double block_max = hipcub::BlockReduce<double, blockSize>(temp_storage).Reduce(local_max, hipcub::Max());

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        errors[blockIdx.y * gridDim.x + blockIdx.x] = block_max;
    }
}


int main(int argc, char const *argv[]) {
    po::options_description desc("options");
    desc.add_options()
        ("eps", po::value<double>()->default_value(1e-6),"Accuracy")
        ("size", po::value<int>()->default_value(10),"Matrix size")
        ("iterations", po::value<int>()->default_value(1000000),"Max count of iteration")
        ("show", po::value<bool>()->default_value(false),"Show ResMatrix")
        ("init", po::value<bool>()->default_value(false),"Use mean value during init")
        ("help", "Show all all command")
    ;

    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << "\n";
        return 1;
    }

    double eps = vm["eps"].as<double>();
    int size = vm["size"].as<int>();
    int iterations = vm["iterations"].as<int>();
    bool showResult = vm["show"].as<bool>();
    bool initMean = vm["init"].as<bool>();

    std::cout << "Current settings:" << std::endl;
    std::cout << "\tEPS: " << eps << std::endl;
    std::cout << "\tMax iteration: " << iterations << std::endl;
    std::cout << "\tSize: " << size << 'x' << size << std::endl;
    std::cout << "\tMean Value: " << initMean << std::endl;

    double error = 1.0;
    int iter = 0;

    Data<double> A(size_sq);
    Data<double> Anew(size_sq);

    initMatrix(A.arr, size);
    initMatrix(Anew.arr, size);

    auto start = std::chrono::high_resolution_clock::now();

    dim3 blockDim(32, 32);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x, (size + blockDim.y - 1) / blockDim.y);

    int totalBlocks = gridDim.x * gridDim.y;
    Data<double> errors(totalBlocks);

    A.copyToDevice();
    Anew.copyToDevice();
    errors.copyToDevice();

    double* A_link = A.getDevicePointer();
    double* Anew_link = Anew.getDevicePointer();
    double* errors_link = errors.getDevicePointer();

    std::unique_ptr<hipStream_t, void(*)(hipStream_t*)> 
    stream(new hipStream_t, [](hipStream_t* s) {
        hipStreamDestroy(*s);
        delete s;
    });

    std::unique_ptr<hipGraph_t, void(*)(hipGraph_t*)> 
    graph(new hipGraph_t, [](hipGraph_t* g) {
        hipGraphDestroy(*g);
        delete g;
    });

    std::unique_ptr<hipGraphExec_t, void(*)(hipGraphExec_t*)> 
    graphExec(new hipGraphExec_t, [](hipGraphExec_t* ge) {
        hipGraphExecDestroy(*ge);
        delete ge;
    });

    hipStreamCreate(stream.get());
    bool graphCreated = false;

    while (iter < iterations && error > eps) {
        if (!graphCreated) 
        {
            hipStreamBeginCapture(*stream, hipStreamCaptureModeGlobal);

            for (int i = 0; i < 999; i++) {
                iterate<<<gridDim, blockDim, 0, *stream>>>(A_link, Anew_link, size);
                std::swap(A_link, Anew_link);
            }

            iterate<<<gridDim, blockDim, 0, *stream>>>(A_link, Anew_link, size);
            compute_error<32><<<gridDim, blockDim, 0, *stream>>>(A_link, Anew_link, errors_link, size);

            hipStreamEndCapture(*stream, graph.get());
            hipGraphInstantiate(graphExec.get(), *graph, nullptr, nullptr, 0);

            graphCreated = true;
        } 
        else 
        {
            hipGraphLaunch(*graphExec, *stream);
            hipStreamSynchronize(*stream);

            errors.copyToHost();
            error = *std::max_element(errors.arr.begin(), errors.arr.end());
            iter += 1000;
        }
    }

    A.copyToHost();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    std::cout << "Iterations: " << iter << "\n";
    std::cout << "Time: " << elapsed.count() << " s\n";
    std::cout << "Error: " << error << "\n";

    saveMatrix(A.arr.data(), size, "result_matrix.txt");

    return 0;
}



// __global__ - вызывается с хоста, запускает функцию на device
// каждый параллельный вызов ф-ции - это block
// набор таких блоков - grid
// block может быть разбит на потоки (threads)
// func <<<N, M>>> - запуск функции на gpu, где N - кол-во блоков, M - кол-во потоков 
// blockDim.x - кол-во потоков в блоке
// __shared__ используется для объявления переменной/массива в общей памяти
// dim3 blockDim(32, 32) - зависит от Warp Size (который у нас 32) группа потоков внутри потоковго блока, 
// которые физически выполняются одновременно